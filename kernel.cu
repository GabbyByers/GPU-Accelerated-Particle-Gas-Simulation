#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"

// simulate
__global__ void simulationKernel(Particle* device_particles, int num_particles) {
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= num_particles)
        return;

    Particle& particle = device_particles[index];
    for (int i = 0; i < num_particles; i++) {
        if (i == index)
            continue;
        Particle& other = device_particles[i];
        particle.simulateGravity(other, 1.0f);
    }
}

void initializeDeviceGravitySimulation(Particle* device_particles, Particle* cpu_particles, int num_particles) {
    hipMemcpy(device_particles, cpu_particles, num_particles * sizeof(Particle), hipMemcpyHostToDevice);
}

void deviceGravitySimulation(Particle* device_particles, int num_particles) {
    int NUM_THREADS = 512;
    int NUM_BLOCKS = (num_particles + NUM_THREADS - 1) / NUM_THREADS; 
    simulationKernel <<<NUM_BLOCKS, NUM_THREADS>>> (device_particles, num_particles);
    hipDeviceSynchronize();
}

// render
__global__ void renderKernel(int* device_pixel_particle_counts, unsigned char* device_image_pixels, int num_pixels) {
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= num_pixels)
        return;

    // determine pixel color
}

void deviceRenderer(int* device_pixel_particle_counts, unsigned char* device_image_pixels, int num_pixels) {
    int NUM_THREADS = 512;
    int NUM_BLOCKS = (num_pixels + NUM_THREADS - 1) / NUM_THREADS;
    renderKernel <<<NUM_BLOCKS, NUM_THREADS>>> (device_pixel_particle_counts, device_image_pixels, num_pixels);
    hipDeviceSynchronize();
}

// malloc
Particle* allocateDeviceParticles(int num_particles) {
    Particle* device_particles = nullptr;
    hipMalloc((void**)&device_particles, num_particles * sizeof(Particle));
    return device_particles;
}

int* allocateDeviceParticlePositions(int num_particles) {
    int* device_particle_positions = nullptr;
    hipMalloc((void**)&device_particle_positions, num_particles * sizeof(int));
    return device_particle_positions;
}

int* allocateDevicePixelParticleCounts(int num_pixels) {
    int* device_pixel_particle_counts = nullptr;
    hipMalloc((void**)&device_pixel_particle_counts, num_pixels * sizeof(int));
    return device_pixel_particle_counts;
}

unsigned char* allocateDeviceImagePixels(int num_pixels) {
    unsigned char* device_image_pixels = nullptr;
    hipMalloc((void**)&device_image_pixels, num_pixels * 4);
    return device_image_pixels;
}

// free
void freeDeviceParticles(Particle* device_particles) {
    hipFree(device_particles);
}

void freeDeviceParticlePositions(int* device_particle_positions) {
    hipFree(device_particle_positions);
}

void freeDevicePixelParticleCounts(int* device_pixel_particle_counts) {
    hipFree(device_pixel_particle_counts);
}

void freeDeviceImagePixels(unsigned char* device_image_pixels) {
    hipFree(device_image_pixels);
}