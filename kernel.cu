#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"

__global__ void kernel(Particle* device_particles, int num_particles)
{
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= num_particles)
        return;

    Particle& particle = device_particles[index];

    for (int i = 0; i < num_particles; i++)
    {
        if (i == index)
            continue;
        Particle& other = device_particles[i];
        particle.repelOther(other.pos, 1.0f);
    }

    particle.repelBox(Vec2f(0.0f, 0.0f), Vec2f(0.4f, 0.4f));

    particle.eulerIntegration();
    particle.repelEdgeScreen();
}

Particle* allocateDeviceMemory(int num_particles)
{
    hipSetDevice(0);
    Particle* device_particles = nullptr;
    hipMalloc((void**)&device_particles, num_particles * sizeof(Particle));
    return device_particles;
}

void runDevice(Particle* particles, Particle* device_particles, int num_particles)
{
    hipMemcpy(device_particles, particles, num_particles * sizeof(Particle), hipMemcpyHostToDevice);

    unsigned int NUM_THREADS = 1024;
    unsigned int NUM_BLOCKS = (num_particles + NUM_THREADS - 1) / NUM_THREADS;
    kernel <<<NUM_BLOCKS, NUM_THREADS>>> (device_particles, num_particles);
    
    hipDeviceSynchronize();
    hipMemcpy(particles, device_particles, num_particles * sizeof(Particle), hipMemcpyDeviceToHost);
}

void freeDeviceMemory(Particle* device_particles)
{
    hipFree(device_particles);
}